#include <pkai/network.hpp>

void PKAI::Network::extract_network_data(float ** _neurons, float ** _synapses, float ** _biases) {
    float * neuron_ptrs[layer_count];
    float * synapse_ptrs[layer_count - 1];
    float * bias_ptrs[layer_count - 1];
    hipMemcpy(neuron_ptrs, neurons, layer_count * sizeof(float *), hipMemcpyDeviceToHost);
    hipMemcpy(synapse_ptrs, synapses, (layer_count - 1) * sizeof(float *), hipMemcpyDeviceToHost);
    hipMemcpy(bias_ptrs, biases, (layer_count - 1) * sizeof(float *), hipMemcpyDeviceToHost);

    for (int i = 0; i < layer_count; i++) {
        hipMemcpy(_neurons[i], neuron_ptrs[i], layer_sizes[i] * sizeof(float), hipMemcpyDeviceToHost);
    }
    for (int i = 0; i < layer_count - 1; i++) {
        hipMemcpy(_synapses[i], synapse_ptrs[i], layer_sizes[i] * layer_sizes[i + 1] * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(_biases[i], bias_ptrs[i], layer_sizes[i + 1] * sizeof(float), hipMemcpyDeviceToHost);
    }
}

void PKAI::Network::send_network_data(float ** _neurons, float ** _synapses, float ** _biases) {
    if (_neurons) {
        float * neuron_ptrs[layer_count];
        hipMemcpy(neuron_ptrs, neurons, layer_count * sizeof(float *), hipMemcpyDeviceToHost);

        for (int i = 0; i < layer_count; i++) {
            hipMemcpy(neuron_ptrs[i], _neurons[i], layer_sizes[i] * sizeof(float), hipMemcpyHostToDevice);
        }
    }

    if (_synapses) {
        float * synapse_ptrs[layer_count - 1];
        hipMemcpy(synapse_ptrs, synapses, (layer_count - 1) * sizeof(float *), hipMemcpyDeviceToHost);

        for (int i = 0; i < layer_count - 1; i++) {
            hipMemcpy(synapse_ptrs[i], _synapses[i], layer_sizes[i] * layer_sizes[i + 1] * sizeof(float),
                       hipMemcpyHostToDevice);
        }
    }

    if (_biases) {
        float * bias_ptrs[layer_count - 1];
        hipMemcpy(bias_ptrs, biases, (layer_count - 1) * sizeof(float *), hipMemcpyDeviceToHost);

        for (int i = 0; i < layer_count - 1; i++) hipMemcpy(
            bias_ptrs[i],
            _biases[i],
            layer_sizes[i + 1] * sizeof(float),
            hipMemcpyHostToDevice
        );
    }
}